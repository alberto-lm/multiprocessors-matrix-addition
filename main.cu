
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <omp.h>

#define runs 5

int rowsA = 1024;
int colsA = 1024;
int rowsB = 1024;
int colsB = 1024;
int numThreads = 8;
int ompCorrect = 1;
int cudaCorrect = 1;
int threadsCorrect = 1;
int i;
int j;
int k;
int count;
int x;

char fileA[] = "matrizA.txt";
char fileB[] = "matrizB.txt";
char fileC[] = "matrizC.txt";

double *a, *b, *c, *r, *h_r, start, end;
double executionTimes[runs + 2][3];
double accumTime, temp, precision = 0.0000000001;

int getMatrixesDimensions() {
	printf("Filas Matriz A = ");
	scanf("%d", &rowsA);
	printf("Columnas Matriz A = ");
	scanf("%d", &colsA);
	printf("Filas Matriz B = ");
	scanf("%d", &rowsB);
	printf("Columnas Matriz B = ");
	scanf("%d", &colsB);
	if (rowsA <= 0 || rowsB <= 0 || colsA <= 0 || colsB <= 0 || colsA != rowsB) {
		printf("La operacion no se puede realizar con las dimensiones especificadas.\n");
		return -1;
	}
	return 0;
}

void writeDataIntoFile() {
	FILE* file = fopen(fileC, "w");
	if (file == NULL) {
		printf("El archivo %s no se crea correctamente.", fileC);
	}
	for (int i = 0; i < rowsA * colsB; i++) {
		fprintf(file, "%5.10lf\n", c[i]);
	}
	fclose(file);
}

int readDataFromFile(double* matrixContainer, int expectedSize, char *fileName) {
	FILE* file = fopen(fileName, "r");
	if (file == NULL) {
		printf("El archivo %s no abria correctamente.", fileName);
		return -3;
	}
	int i = 0;
	while (i < expectedSize && fscanf(file, "%lf", &matrixContainer[i]) != EOF){	
		i++;
	}
	printf("\n");
	fclose(file);
	if (i < expectedSize) {
		printf("La cantidad de elementos leidos del archivo %s no permite construir la matriz especificada.", fileName);
		return -3;
	}
	return 0;
}

int buildMatrixes() {
	size_t aSize, bSize, outSize;
	aSize = rowsA * colsA * sizeof(double);
	bSize = rowsB * colsB * sizeof(double);
	outSize = rowsA * colsB * sizeof(double);
	a = (double*)malloc(aSize);
	b = (double*)malloc(bSize);
	c = (double*)malloc(outSize);
	r = (double*)malloc(outSize);
	h_r = (double*)malloc(outSize);
	if (errno == ENOMEM || a == NULL || b == NULL || c == NULL) {
		printf("Error en memory allocation o especio insuficiente en el heap.\n");
		return -2;
	}
	return 0;
}

void freeMemory() {
	free(a);
	free(b);
	free(c);
	free(r);
	free(h_r);
}

int verifyResults(double *outputMatrix) {
	for (i = 0; i < rowsA * colsB; i++) {
		if (abs(c[i] - outputMatrix[i]) > precision) {
			printf("Wroonggg!!\n");
			printf("c = %.10lf\tout = %.10lf\n", c[i], outputMatrix[i]);
			return 0;
		}
	}
	return 1;
}

void serialMul() {
	accumTime = 0;
	for (x = 0; x < runs; x++) {
		start = omp_get_wtime();
		for (i = 0; i < rowsA; i++) {
			for (j = 0; j < colsB; j++) {
				c[i * colsB + j] = 0;
				for (k = 0; k < colsA; k++) {
					c[i * colsB + j] += a[i * colsA + k] * b[k * colsB + j];;
				}
			}
		}
		end = omp_get_wtime();
		accumTime += (end - start);
		executionTimes[x][0] = (end - start) * 1000;
	}
	executionTimes[runs + 1][0] = accumTime / runs  * 1000;
	writeDataIntoFile();
	printf("OMP finished\n");

}

void parallelOMP() {
	accumTime = 0;
	omp_set_num_threads(numThreads);
	for (x = 0; x < runs; x++) {
		#pragma omp parallel shared (a, b, c, r) private (count, i, j, k)
		{
			start = omp_get_wtime();
				#pragma omp for
				for (count = 0; count < rowsA * colsB; count++) {
						i = count / colsB;
						j = count % colsB;
						r[count] = 0;
						for (k = 0; k < colsA; k++) {
							r[count] += a[i * colsA + k] * b[k * colsB + j];;
						}
					
				}
			}
			end = omp_get_wtime();
		  	accumTime += (end - start);
		  	executionTimes[x][1] = (end - start) * 1000;
	}
	executionTimes[runs + 1][1] = accumTime / runs * 1000;
	ompCorrect = verifyResults(r);
	printf("OMP finished\n");

}

__global__ void cudaMul(int *cudaIterations, double * d_a, double * d_b, double * d_r){
	double accum = 0.0;
	int n = *cudaIterations;
	for (int i = 0; i < n; i++){
		//printf("blockIdx = %d\tthreadIdx = %d\ta = %lf\tb = %lf\n", blockIdx.x, threadIdx.x, a[blockIdx.x * n + i], b[i * n + threadIdx.x]);
		accum += d_a[blockIdx.x * n + i] * d_b[i * n + threadIdx.x];
	}
	d_r[blockIdx.x * n + threadIdx.x] = accum;
}

void cudaWrapper(){
	accumTime = 0;
	int *cudaIterations;
	double *d_r, *d_a, *d_b;
	hipMalloc((int**)& cudaIterations, sizeof(int));
	hipMemcpy(cudaIterations, &colsA, sizeof(int) , hipMemcpyHostToDevice);
	hipMalloc((double**)& d_a, rowsA * colsA * sizeof(double));
	hipMalloc((double**)& d_b, rowsB * colsB * sizeof(double));
	hipMalloc((double**)& d_r, rowsA * colsB * sizeof(double));
	hipMemcpy(d_a, a, rowsA * colsA * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, rowsB * colsB * sizeof(double), hipMemcpyHostToDevice);
	for (x = 0; x < runs; x++) {
		start = omp_get_wtime();
		cudaMul <<< rowsA, colsB >>> (cudaIterations, d_a, d_b, d_r);
		hipDeviceSynchronize();
		end = omp_get_wtime();
		accumTime += (end - start);
		executionTimes[x][2] = (end - start) * 1000;
	}
	hipMemcpy(h_r, d_r, rowsA * colsB * sizeof(double), hipMemcpyDeviceToHost);
	executionTimes[runs + 1][2] = accumTime / runs * 1000;
	cudaCorrect = verifyResults(h_r);
	hipFree(cudaIterations);
	hipFree(d_r);
	hipFree(d_a);
	hipFree(d_b);
}

void printTable() {
	printf("%-20s%-20s%-20s%-20s\n","Corrida", "Serial", "Paralelo1", "Paralelo2");
	for (i = 0; i < runs; i++) {
		printf ("%-20d%-20.10lf%-20.10lf%-20.10lf\n", i+1, executionTimes[i][0], executionTimes[i][1], executionTimes[i][2]);
	}
	printf ("%-20s%-20.10lf%-20.10lf%-20.10lf\n", "Promedio", executionTimes[runs + 1][0], executionTimes[runs + 1][1], executionTimes[runs + 1][2]);
	printf ("%-20s%-20s%-20.10lf%-20.10lf\n", "Speedup", "-",  executionTimes[runs + 1][1] / executionTimes[runs + 1][0], executionTimes[runs + 1][2] / executionTimes[runs + 1][0]);
	printf ("%-20s%-20s%-20d%-20d\n", "Correct", "-", ompCorrect, cudaCorrect);
}

int main() {
	/*if (getMatrixesDimensions()) {
		return -1;
	}*/
	if (buildMatrixes()) {
		return -2;
	}
	if (readDataFromFile(a, rowsA * colsA, fileA) || readDataFromFile(b, rowsB * colsB, fileB)) {
		freeMemory();
		return -3;
	}
	serialMul();
	parallelOMP();
	cudaWrapper();
	printTable();
	freeMemory();
	return 0;
}
